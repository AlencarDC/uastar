#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <moderngpu.cuh>

template<int NT, int VT, typename Key>
__global__ void BlockSortKernel(Key *d_in, Key *d_out)
{
    int tid = threadIdx.x;
    Key key[VT];

    __shared__ Key shared[NT*(VT+1)];

    mgpu::DeviceGlobalToThread<NT, VT>(NT*VT, d_in, tid, key);
    mgpu::CTAMergesortKeys<NT, VT, false>(key, shared, VT*NT, tid, mgpu::less<int>());
    mgpu::DeviceSharedToGlobal<NT, VT>(NT*VT, shared, tid, d_out);
}

int main(int argc, char *argv[])
{
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(argc, argv, true);

    MGPU_MEM(int32_t) d_in  = context->GenRandom<int32_t>(128*3, 1, 100);
    MGPU_MEM(int32_t) d_out = context->Malloc<int32_t>(128*3);

    context->Start();
    BlockSortKernel<128, 3, int32_t><<<1, 128>>>(*d_in, *d_out);
    double elapsed = context->Split();

    printf("Time elapsed: %.2f\n", elapsed);

    puts("Input array: ");
    mgpu::PrintArray(*d_in, "%6d", 10);
    puts("Output array: ");
    mgpu::PrintArray(*d_out, "%6d", 10);

    return 0;
}
